
#include <hip/hip_runtime.h>
//******************************************************************************
//
// File:    MetricCenterGpu.cu
// Unit:    MetricCenterGpu kernel function
//
// This C/CUDA source file is copyright (C) 2014 by Utkarsh Bhatia. 
// This class is extending Task class as given in the PJ2 parallel java library
// made by Professor Alan Kaminsky, the given reference to parallel java library and its sample
// code can be referenced from http://www.cs.rit.edu/~ark/bcbd/#source and http://www.cs.rit.edu/~ark/pj2.shtml
// This class is used for running the program with MetricCenterGpu.cubin for running the
// program in CUDA which works in multiple GPU cores.
//
// Details for PJ2 library as available on http://www.cs.rit.edu/~ark/pj2.shtml
// The library has been made available to General Public under GPL license by 
// Professor Alan Kaminsky. The copyright (C) 2015 to pj2 library is held by Alan Kaminsky.
// PJ2 is free software; you can redistribute it and/or modify it under the terms of
// the GNU General Public License as published by the Free Software Foundation;
// either version 3 of the License, or (at your option) any later version.
//
// PJ2 is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// A copy of the GNU General Public License is provided in the file gpl.txt. You
// may also obtain a copy of the GNU General Public License on the World Wide
// Web at http://www.gnu.org/licenses/gpl.html.
//
// Created on: Nov 6, 2015
// 
// Author: Utkarsh Bhatia
// 
//******************************************************************************

// Number of threads per block.
#define NT 1024

// Structure for a 2-D vector.
typedef struct
{
 double x;
 double y;
}
VectorInput;

// Structure for a radius and index vector
typedef struct
{
 double r;
 int i;
}
VectorResult;

// Per-thread variables in shared memory.
__shared__ VectorResult resultArray[NT];

/*Calculate temperory distance for the current point being processed in the current thread
 *
 * @param  cord1 cord1 to calculate distance
 * @param  cord2 cord2 to calculate distance
 *
 * @return
 * 	returns distance from cord1 to cord2
 */
__device__ double tempDistance(VectorInput *cord1, VectorInput *cord2)
{
	double diffX = cord1->x - cord2->x;
	double diffY = cord1->y - cord2->y;
	double tempDist = sqrt( pow(diffX, 2) + pow(diffY, 2) );
	return tempDist;
}

/*get maximum distance for current selected point at block level
 *
 * @param  oldMax current maximum value from the selected point
 * @param  newDist calculated new distance from the selected point
 *
 * @return
 * 	returns new maximum distance
 */
__device__ double getMaxDist(double oldMax, double newDist)
{
	if(newDist > oldMax){
		return newDist;
	}
	return oldMax;
}

/*set new values via reduction on the thread level
 *
 * @param  oldVal current maximum radius from the selected point
 * @param  newVal calculated new radius for the current selected point
 *
 */
__device__ void setNewValues(VectorResult *oldVal, VectorResult *newVal)
{
	if(oldVal->r < newVal->r){
		oldVal->i = newVal->i;
		oldVal->r = newVal->r;
	}
}

//reducing the result on the block level
/*set new values via reduction on the thread level
 *
 * @param  currR current maximum radius from the selected point
 * @param  finalVal calculated new radius for the next selected point
 *
 */
__device__ void resultReduction(VectorResult *currR, VectorResult *finalVal)
{
	if(currR->r == -1){
		currR->r = finalVal->r;
		currR->i = finalVal->i;
	} else{
		if(currR->r > finalVal->r){
			currR->r = finalVal->r;
			currR->i = finalVal->i;
		}
	}
}

/**
 * Device kernel to calculate metric center on the given input.
 * <P>
 * Called with a one-dimensional grid of one-dimensional blocks, Each block updates one values
 * current chosen point. The points are balanced across all given blocks. Each
 * thread within a block computes the distance with respect to the set cordinate in the block
 *
 * @param  pList  Array of input point list.
 * @param  rList  Array of result list storing all metric centers of each blocks.
 * @param  N     Number of input cordinates
 *
 * @author  Utkarsh Bhatia
 * @version 06-Nov-2015
 */
extern "C" __global__ void metricCalculate
(VectorInput *pList, VectorResult *rList, int N)
{
	int totalBlocks = gridDim.x; //total number of blocks
	int idBlock = blockIdx.x; //id of the current block
	int idThread = threadIdx.x; //id of the current thread

	//current tempIndex being processed in the block
	double tempIndex = 0;
	//dist of the current two points
	double dist = 0;
	//max distance of the current point in the block
	double maxDist=0.0;

	//compute and calculate distance of the the point w.r.t to every other cordinate
	for(int i = idBlock; i < N; i = i + totalBlocks) {
		tempIndex = i;
		maxDist=0.0;
		for(int j = idThread; j < N; j = j + NT) {
			dist = tempDistance(&pList[i], &pList[j]);
			maxDist = getMaxDist(maxDist, dist);
		
		}

		resultArray[idThread] = (VectorResult){maxDist,tempIndex};


		// Compute largest distance via shared memory parallel reduction.
		__syncthreads();
		for (int k = NT/2; k > 0; k >>= 1)
		{
			if (idThread < k)
			{
				VectorResult *temp1 = &resultArray[idThread];
				VectorResult *temp2 = &resultArray[idThread+k];
				//calling method to reduce the thread level reduction
				setNewValues(temp1, temp2);
			}
			__syncthreads();
		}

		// Single threaded section.
		if (idThread == 0)
		{
			//calling method to reduce the thread level reduction
			resultReduction(&rList[idBlock], &resultArray[0]);

		}	
	}
}

